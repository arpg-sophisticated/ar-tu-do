// compile with nvcc matrix_addition.cu -o matrix_addition
// execute using ./matrix_addition


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x*threadIdx.x+threadIdx.x] = a[blockIdx.x*threadIdx.x+threadIdx.x] + b[blockIdx.x*threadIdx.x+threadIdx.x];
    printf("Block %d Thread %d -- a: %d, b: %d, c: %d\n", blockIdx.x, threadIdx.x, c[blockIdx.x*threadIdx.x+threadIdx.x], a[blockIdx.x*threadIdx.x+threadIdx.x], b[blockIdx.x*threadIdx.x+threadIdx.x]);
}

void random_ints(int* a, int N, int M) {
    int i, j;
    for (i = 0; i < N; ++i) {
        for (j = 0; j < M; ++j) {
            a[i*j+j] = rand();
        }
    }
}

// 4096 values in total
#define N 64
#define M 64
int main(void) {
    int *a, *b, *c;         // host copies of a, b, c
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int size = N * M * sizeof(int);

    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N, M);
    b = (int *)malloc(size); random_ints(b, N, M);
    c = (int *)malloc(size);
    
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernel on GPU with N blocks
    add<<<N,M>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}